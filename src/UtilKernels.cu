#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>

namespace GPU {

template <typename Scalar>
__global__ void permuteKernel(const Scalar* __restrict__ v, const int* __restrict__ perm, Scalar* __restrict__ dst, int size)
{
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    dst[perm[i]] = v[i];
  }
}

template <typename Scalar>
__global__ void addInPlaceKernel(Scalar * __restrict__ lhs, Scalar * __restrict__ rhs)
{
  *lhs += *rhs;
}

template <>
__global__ void addInPlaceKernel<hipComplex>(hipComplex *__restrict__ a, hipComplex *__restrict__ b)
{
  a->x += b->x;
  a->y += b->y;
}

template <>
__global__ void addInPlaceKernel<hipDoubleComplex>(hipDoubleComplex *__restrict__ a, hipDoubleComplex *__restrict__ b)
{
  a->x += b->x;
  a->y += b->y;
}

template <typename Scalar, typename RealType>
__global__ void divideByRealKernel(Scalar * __restrict__ v, const RealType * __restrict__ s, Scalar * __restrict__ res, int size) {
  RealType scalar = *s;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    v[i] /= scalar;
  }
}

template <>
__global__ void divideByRealKernel<hipComplex, float>(hipComplex * __restrict__ v, const float * __restrict__ s, hipComplex * __restrict__ res, int size) {
  float scalar = *s;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    res[i].x = v[i].x / scalar;
    res[i].y = v[i].y / scalar;
  }
}

template <>
__global__ void divideByRealKernel<hipDoubleComplex, double>(hipDoubleComplex * __restrict__ v, const double * __restrict__ s, hipDoubleComplex * __restrict__ res, int size) {
  double scalar = *s;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    res[i].x = v[i].x / scalar;
    res[i].y = v[i].y / scalar;
  }
}


template <typename Scalar>
__global__ void eigshNormalizeKernel(Scalar * __restrict__ col, Scalar * __restrict__ v, int n,
                                     const Scalar * __restrict__ u ,const Scalar * __restrict__ beta)
{
  Scalar b = *beta;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    Scalar res = u[i] / b;
    col[i] = res;
    v[i] = res;
  }
}

template <>
__global__ void eigshNormalizeKernel<hipComplex>(hipComplex * __restrict__ col, hipComplex * __restrict__ v, int n,
                                                const hipComplex * __restrict__ u ,const hipComplex * __restrict__ beta)
{
  float b = beta->x;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    hipComplex res = make_hipComplex(u[i].x / b, u[i].y / b);
    col[i] = res;
    v[i] = res;
  }
}

template <>
__global__ void eigshNormalizeKernel<hipDoubleComplex>(hipDoubleComplex * __restrict__ col, hipDoubleComplex * __restrict__ v, int n,
                                                      const hipDoubleComplex * __restrict__ u ,const hipDoubleComplex * __restrict__ beta)
{
  double b = beta->x;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    hipDoubleComplex res = make_hipDoubleComplex(u[i].x / b, u[i].y / b);
    col[i] = res;
    v[i] = res;
  }
}

template <typename Scalar>
void _permute(Scalar *v, const int *perm, void *buffer, int size)
{
  permuteKernel<Scalar><<<std::min((int) std::ceil(size / 512.0f), 80), 512>>>(v, perm, (Scalar *) buffer, size);
  CHECK_CUDA( hipMemcpy(v, buffer, size * sizeof(Scalar), hipMemcpyDeviceToDevice) );
}

template <typename Scalar>
void addInPlace(Scalar * __restrict__ lhs, Scalar * __restrict__ rhs) {
  addInPlaceKernel<Scalar><<<1, 1>>>(lhs, rhs);
}

template <typename Scalar>
inline void eigshNormalize(Scalar * __restrict__ col, Scalar * __restrict__ v, int n,
                           const Scalar * __restrict__ u ,const Scalar * __restrict__ beta)
{
  eigshNormalizeKernel<Scalar><<<std::min((int) std::ceil(n / 512.0f), 80), 512>>>(col, v, n, u, beta);
}

template <typename Scalar, typename RealType>
void _divideByReal(Scalar * __restrict__ v, const RealType * __restrict__ s, Scalar * __restrict__ res, int size) {
  divideByRealKernel<Scalar, RealType><<<std::min((int) std::ceil(n / 512.0f), 80), 512>>>(v, s, res, size);
}

} // Namespace GPU
